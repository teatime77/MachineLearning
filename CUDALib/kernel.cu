
#include "hip/hip_runtime.h"


#include <stdio.h>

struct ArrayN {
	double* dt;
	double* DevDt;
	int Length;

	void Alloc(){
		hipError_t  cudaStatus = hipMalloc((void**)&DevDt, Length * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw -1;
		}
	}

	void Free(){
		hipFree(DevDt);
	}

	void ToDev(){
		if (DevDt == NULL){
			Alloc();
		}

		hipError_t cudaStatus = hipMemcpy(DevDt, dt, Length * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw - 1;
		}
	}

	void ToHost(){
		hipError_t cudaStatus = hipMemcpy(dt, DevDt, Length * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw - 1;
		}
	}
};

struct Array1 : ArrayN {
};

struct Array2 : ArrayN {
	int nRow;
	int nCol;
};

__global__ void addKernel(Array1 c, const Array1 a, const Array1 b)
{
    int i = threadIdx.x;
	c.DevDt[i] = a.DevDt[i] + b.DevDt[i];
}


extern "C" __declspec(dllexport) int CUDASetDevice(int device){
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	return (int)hipSuccess;
}

extern "C" __declspec(dllexport) int CUDADeviceReset(){
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return (int)hipSuccess;
}

// Helper function for using CUDA to add vectors in parallel.
extern "C" __declspec(dllexport) int addWithCuda(Array1 a, Array1 b, Array1 c)
{
    hipError_t cudaStatus;

	a.ToDev();
	b.ToDev();
	c.Alloc();
	hipDeviceSynchronize();

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, c.Length>>>(c, a, b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	c.ToHost();
	hipDeviceSynchronize();

Error:
	c.Free();
	a.Free();
	b.Free();
    
    return (int)cudaStatus;
}
