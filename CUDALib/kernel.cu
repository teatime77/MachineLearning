
#include "hip/hip_runtime.h"


#include <stdio.h>

struct ArrayN {

};

struct Array1 : ArrayN {
	double* dt;
	double* DevDt;
	int Length;

	void Alloc(){
		hipError_t  cudaStatus = hipMalloc((void**)&DevDt, Length * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw -1;
		}
	}

	void Free(){
		hipFree(DevDt);
	}

	void ToDev(){
		hipError_t cudaStatus = hipMemcpy(DevDt, dt, Length * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw - 1;
		}
	}

	void ToHost(){
		hipError_t cudaStatus = hipMemcpy(dt, DevDt, Length * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw - 1;
		}
	}
};

__global__ void addKernel(Array1 c, const Array1 a, const Array1 b)
{
    int i = threadIdx.x;
	c.DevDt[i] = a.DevDt[i] + b.DevDt[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(Array1 c, Array1 a, Array1 b, unsigned int size)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output)    .
	c.Alloc();
	a.Alloc();
	b.Alloc();

	// Copy input vectors from host memory to GPU buffers.
	a.ToDev();
	b.ToDev();
	

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(c, a, b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	c.ToHost();

Error:
	c.Free();
	a.Free();
	b.Free();
    
    return cudaStatus;
}

extern "C" __declspec(dllexport) int CUDAmain(Array1 a, Array1 b, Array1 c)
{
	const int arraySize = 5;
	//const double a[arraySize] = { 1, 2, 3, 4, 5 };
	//const double b[arraySize] = { 10, 20, 30, 40, 50 };
	//double c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
	//    c[0], c[1], c[2], c[3], c[4]);

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}
